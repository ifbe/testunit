#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void increment_kernel(float* g_data)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	g_data[idx] = idx*1.0;
}

void printoutput(float* data, int len)
{
	if(len>48)len=48;
	for (int i = 0; i < len; i++){
		printf("%4.1f%c", data[i], ((i&0xf)==15)?'\n':' ');
	}
}
int main(int argc, char *argv[])
{
	hipSetDevice(0);

	int n = 16 * 1024 * 1024;
	int nbytes = n * sizeof(float);

	dim3 threads = dim3(512, 1, 1);
	dim3 blocks  = dim3(n / threads.x, 1, 1);

	// allocate host memory
	float *cpumem = 0;
	hipHostMalloc((void **)&cpumem, nbytes, hipHostMallocDefault);
	memset(cpumem, 0, nbytes);

	// allocate device memory
	float *gpumem = 0;
	hipMalloc((void **)&gpumem, nbytes);
	hipMemset(gpumem, 255, nbytes);

	// create cuda event handles
	hipEvent_t event[4];
	for(int i=0;i<4;i++)hipEventCreate(&event[i]);

	hipDeviceSynchronize();

	// asynchronously issue work to the GPU (all to stream 0)
	hipEventRecord(event[0], 0);
	hipMemcpyAsync(gpumem, cpumem, nbytes, hipMemcpyHostToDevice, 0);
	hipEventRecord(event[1], 0);
	increment_kernel<<<blocks, threads, 0, 0>>>(gpumem);
	hipEventRecord(event[2], 0);
	hipMemcpyAsync(cpumem, gpumem, nbytes, hipMemcpyDeviceToHost, 0);
	hipEventRecord(event[3], 0);

	// have CPU do some work while waiting for stage 1 to finish
	unsigned long int counter=0;
	while (hipEventQuery(event[3]) == hipErrorNotReady)
	{
		counter++;
	}

	float gputime[3] = {};
	for(int i=0;i<3;i++)hipEventElapsedTime(&gputime[i], event[i], event[i+1]);
	printf("time spent executing by the GPU: %.f, %f, %f\n", gputime[0], gputime[1], gputime[2]);

	printf("cycle spent executing by the CPU: %lu\n", counter);

	printoutput(cpumem, n);

	// release resources
	for(int i=0;i<4;i++)hipEventDestroy(event[i]);
	hipHostFree(cpumem);
	hipFree(gpumem);
}
