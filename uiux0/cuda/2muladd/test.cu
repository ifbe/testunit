#include <stdio.h>
#include <hip/hip_runtime.h>

#define u64 unsigned long long
#ifdef _WIN32
#include <windows.h>
u64 time_in_ns()
{
	LARGE_INTEGER count,freq;
	int ret = QueryPerformanceFrequency(&freq);
	if(ret && freq.QuadPart){
		ret = QueryPerformanceCounter(&count);
		//say("count=%lld,freq=%lld,time=%lld\n", count.QuadPart, freq.QuadPart, (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000));
		if(ret && count.QuadPart)return (u64)count.QuadPart*1000*1000 / (freq.QuadPart/1000);		//without (u64)=overflow, 10^9*count/freq = overflow
	}

	return 1000 * 1000 * GetTickCount64();
}
#elif __APPLE__
#include <mach/mach_time.h>
#define lseek64 lseek
u64 time_in_ns()
{
	return mach_absolute_time();
}
#else
#include <time.h>
u64 time_in_ns()
{
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (u64)t.tv_sec*1000*1000*1000 + t.tv_nsec;
}
#endif

__global__ void muladd_kernel(float* out, float* vec, float* mat, int xdim, int ydim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	float f = 0.0;
	for(x=0;x<xdim;x++)f += mat[idx*xdim + x] * vec[x];
	out[idx] = f;
}

void printoutput(float* data, int len)
{
	int y;
	printf("[%d,%d)\n", 0, 31);
	for(y=0;y<2;y++){
		printf("%.1f, %.1f, %.1f, %.1f......%.1f, %.1f\n", data[y*16+0], data[y*16+1], data[y*16+2], data[y*16+3], data[y*16+14], data[y*16+15]);
	}
	printf("[%d,%d)\n", 16*254, 16*256-1);
	for(y=254;y<256;y++){
		printf("%.1f, %.1f, %.1f, %.1f......%.1f, %.1f\n", data[y*16+0], data[y*16+1], data[y*16+2], data[y*16+3], data[y*16+14], data[y*16+15]);
	}
}




static int xdim = 16384;
static int ydim = 32000;
static int vecbyte = xdim * sizeof(float);
static int matbyte = xdim * ydim * sizeof(float);
static float *cpuout = 0;
static float *cpuvec = 0;
static float *cpumat = 0;
static float *gpuout = 0;
static float *gpuvec = 0;
static float *gpumat = 0;
void compute()
{
	xdim = 4096;
	ydim = 4096;
	vecbyte = xdim * sizeof(float);
	matbyte = xdim * ydim * sizeof(float);
	dim3 threads = dim3(512, 1, 1);
	dim3 blocks  = dim3(ydim/512, 1, 1);

	int x,y;
	for(y=0;y<ydim;y++){
		for(x=0;x<xdim;x++){
				cpumat[y*xdim + x] = (x==y) ? 1.0 : 0.0;
		}
		cpumat[y*xdim+17] += 100*1000.0;
	}
	for(x=0;x<xdim;x++){
		cpuvec[x] = x*1.0;
	}

	// create cuda event handles
	u64 time[4];
	hipEvent_t event[4];
	for(int i=0;i<4;i++)hipEventCreate(&event[i]);

	hipDeviceSynchronize();

	// asynchronously issue work to the GPU (all to stream 0)
	time[0] = time_in_ns();
	hipEventRecord(event[0], 0);
	hipMemcpyAsync(gpuvec, cpuvec, vecbyte, hipMemcpyHostToDevice, 0);
	hipMemcpyAsync(gpumat, cpumat, matbyte, hipMemcpyHostToDevice, 0);
	hipEventRecord(event[1], 0);
	muladd_kernel<<<blocks, threads, 0, 0>>>(gpuout, gpuvec, gpumat, xdim, ydim);
	hipEventRecord(event[2], 0);
	hipMemcpyAsync(cpuout, gpuout, vecbyte, hipMemcpyDeviceToHost, 0);
	hipEventRecord(event[3], 0);
	time[1] = time_in_ns();

	// have CPU do some work while waiting for stage 1 to finish
	unsigned long int counter=0;
	while (hipEventQuery(event[3]) == hipErrorNotReady)
	{
		counter++;
	}
	time[2] = time_in_ns();

	float gputime[3] = {};
	for(int i=0;i<3;i++)hipEventElapsedTime(&gputime[i], event[i], event[i+1]);
	printf("time spent executing by the GPU: %f, %f, %f\n", gputime[0], gputime[1], gputime[2]);

	printf("time spent executing by the CPU: %f, %f\n", (time[1]-time[0])*1e-6, (time[2]-time[1])*1e-6);

	printf("cycle spent executing by the CPU: %lu\n", counter);

	printoutput(cpuout, ydim);

	for(int i=0;i<4;i++)hipEventDestroy(event[i]);
}




void backend_init()
{
	u64 t0 = time_in_ns();
	hipSetDevice(0);

	// allocate host memory
	hipHostMalloc((void **)&cpuout, vecbyte, hipHostMallocDefault);
	hipHostMalloc((void **)&cpuvec, vecbyte, hipHostMallocDefault);
	hipHostMalloc((void **)&cpumat, matbyte, hipHostMallocDefault);

	// allocate device memory
	hipMalloc((void **)&gpuout, vecbyte);
	hipMalloc((void **)&gpuvec, vecbyte);
	hipMalloc((void **)&gpumat, matbyte);
	//cudaMemset(gpumem, 255, nbytes);

	u64 t1 = time_in_ns();
	printf("backend_init costtime: %f\n", (t1-t0)*1e-6);
}
void backend_exit()
{
	u64 t0 = time_in_ns();

	hipFree(gpumat);
	hipFree(gpuvec);
	hipFree(gpuout);
	hipHostFree(cpumat);
	hipHostFree(cpuvec);
	hipHostFree(cpuout);

	u64 t1 = time_in_ns();
	printf("backend_exit costtime: %f\n", (t1-t0)*1e-6);
}
int main(int argc, char *argv[])
{
	backend_init();
	compute();
	backend_exit();
}
