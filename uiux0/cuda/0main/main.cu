
#include <hip/hip_runtime.h>
#include<stdio.h>
#define DEV 0
/*  A sample program for nvcc compile */


//device code
__global__ void helloFromGPU()
{
    printf("Hello World from GPU...\n");
}

int main()
{
    printf("Hello World from CPU\n");    //host code

    hipSetDevice(0);                   //device code
    helloFromGPU<<<1,10>>>();           //device code
    hipDeviceReset();                  //device code
    return 0;
}
