#include "hip/hip_runtime.h"
#include<stdio.h>


__global__ void helloFromGPU()
{
    printf("Hello World from GPU...\n");
}

int main()
{
	int count = 0;
	int ret = hipGetDeviceCount(&count);
	printf("hipGetDeviceCount:count=%d,ret=%d\n", count, ret);
	if(count<=0)return 0;

	int j;
	char tmpstr[32];
	hipDeviceProp_t prop;
	for(j=0;j<count;j++){
		ret = hipGetDeviceProperties(&prop, j);
		printf("hipGetDeviceProperties %d\n", j);
		printf("	name: [%s]\n", prop.name);
		printf("	totalGlobalMem: [%zx](%zdMB)\n", prop.totalGlobalMem, prop.totalGlobalMem/1024/1024);
		printf("	sharedMemPerBlock: [%zd]\n", prop.sharedMemPerBlock);
		printf("	regsPerBlock: [%d]\n", prop.regsPerBlock);
		printf("	warpSize: [%d]\n", prop.warpSize);
		printf("	memPitch: [%zd]\n", prop.memPitch);
		printf("	maxThreadsPerBlock: [%d]\n", prop.maxThreadsPerBlock);
		printf("	maxThreadsDim: [%d,%d,%d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("	maxGridSize: [%d,%d,%d]\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("	totalConstMem: [%zd]\n", prop.totalConstMem);
		printf("	version: [%d,%d]\n", prop.major, prop.minor);
		printf("	clockRate: [%d]\n", prop.clockRate);
		printf("	textureAlignment: [%zd]\n", prop.textureAlignment);
		printf("	deviceOverlap: [%d]\n", prop.deviceOverlap);
		printf("	multiProcessorCount: [%d]\n", prop.multiProcessorCount);
		printf("	kernelExecTimeoutEnabled: [%d]\n", prop.kernelExecTimeoutEnabled);
		printf("	integrated: [%d]\n", prop.integrated);
		printf("	canMapHostMemory: [%d]\n", prop.canMapHostMemory);
		printf("	computeMode: [%d]\n", prop.computeMode);
		printf("	maxTexture1D: [%d]\n", prop.maxTexture1D);
		printf("	maxTexture2D: [%d,%d]\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);
		printf("	maxTexture3D: [%d,%d,%d]\n", prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2]);
		printf("	concurrentKernels: [%d]\n", prop.concurrentKernels);

		hipDeviceGetPCIBusId(tmpstr, 32, j);
		printf("	pcibus: [%s]\n", tmpstr);

		printf("\n");
	}

	hipSetDevice(0);

	size_t avail,total;
	hipMemGetInfo(&avail, &total);
	printf("hipMemGetInfo: avail=%zdMB, total=%zdMB\n", avail/1024/1024, total/1024/1024);
	printf("\n");

	helloFromGPU<<<1,10>>>();

	hipDeviceReset();
	return 0;
}
